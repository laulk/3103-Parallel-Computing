//blur cuda
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>

#define TY 32
#define TX 32


//this kernel does the regular blurring process
__global__
void blurKernel (int *R, int *G, int *B, int *Rnew, int *Gnew, int *Bnew, int rowsize, int colsize)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;
	int x = (row * colsize) + col;
	int xu = x - colsize;
	int xd = x + colsize;
	int xr = x + 1;
	int xl = x - 1;

	if((row < rowsize) && (col < colsize)) {
		if (row != 0 && row != (rowsize-1) && col != 0 && col != (colsize-1)){
			Rnew[x] = (R[xr]+R[xl]+R[xd]+R[xu])/4;
			Gnew[x] = (G[xr]+G[xl]+G[xd]+G[xu])/4;
			Bnew[x] = (B[xr]+B[xl]+B[xd]+B[xu])/4;
		}
		else if (row == 0 && col != 0 && col != (colsize-1)){
			Rnew[x] = (R[xr]+R[xd]+R[xl])/3;
			Gnew[x] = (G[xr]+G[xd]+G[xl])/3;
			Bnew[x] = (B[xr]+B[xd]+B[xl])/3;
		}
		else if (row == (rowsize-1) && col != 0 && col != (colsize-1)){
			Rnew[x] = (R[xl]+R[xr]+R[xu])/3;
			Gnew[x] = (G[xl]+G[xr]+G[xu])/3;
			Bnew[x] = (B[xl]+B[xr]+B[xu])/3;
		}
		else if (col == 0 && row != 0 && row != (rowsize-1)){
			Rnew[x] = (R[xr]+R[xu]+R[xd])/3;
			Gnew[x] = (G[xr]+G[xu]+G[xd])/3;
			Bnew[x] = (B[xr]+B[xu]+B[xd])/3;
		}
		else if (col == (colsize-1) && row != 0 && row != (rowsize-1)){
			Rnew[x] = (R[xd]+R[xl]+R[xu])/3;
			Gnew[x] = (G[xd]+G[xl]+G[xu])/3;
			Bnew[x] = (B[xd]+B[xl]+B[xu])/3;
		}
		else if (row==0 &&col==0){
			Rnew[x] = (R[xd]+R[xr])/2;
			Gnew[x] = (G[xd]+G[xr])/2;
			Bnew[x] = (B[xd]+B[xr])/2;
		}
		else if (row==0 &&col==(colsize-1)){
			Rnew[x] = (R[xd]+R[xl])/2;
			Gnew[x] = (G[xd]+G[xl])/2;
			Bnew[x] = (B[xd]+B[xl])/2;
		}
		else if (row==(rowsize-1) &&col==0){
			Rnew[x] = (R[xu]+R[xr])/2;
			Gnew[x] = (G[xu]+G[xr])/2;
			Bnew[x] = (B[xu]+B[xr])/2;
		}
		else if (row==(rowsize-1) &&col==(colsize-1)){
			Rnew[x] = (R[xu]+R[xl])/2;
			Gnew[x] = (G[xu]+G[xl])/2;
			Bnew[x] = (B[xu]+B[xl])/2;
		}
	}
}


__global__
void copyKernel(int *R, int *B, int *G, int *Rnew, int *Gnew, int *Bnew, int rowsize, int colsize)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;
	int x = (row * colsize) + col;

	if((col < colsize) && (row < rowsize)){
		R[x] = Rnew[x];
		G[x] = Gnew[x];
		B[x] = Bnew[x];
	}
}

int main (int argc, const char * argv[]) {
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int R[rowsize][colsize], G[rowsize][colsize], B[rowsize][colsize];
	int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;
	
	// 5a. timing reading of image

	gettimeofday(&tim, NULL);
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);
	fp = fopen ("David.ps", "r");

	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);
				
				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					R[row][col] = h1;
					G[row][col] = h2;
					B[row][col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);

	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("part 5a Reading Image File took :%.6lf seconds elapsed\n", t2-t1);
	//printf("%.6lf,", t2-t1 ); easy csv output

	nblurs = 160;

	//5b :timing allocation of device memory
	gettimeofday(&tim, NULL);
	double t1b=tim.tv_sec+(tim.tv_usec/1000000.0);

	//intialise int variables
	int *d_R;
	int *d_G;
	int *d_B;

	int *d_Rnew , *d_Gnew , *d_Bnew;

	int sizei = sizeof(int) * (rowsize*colsize);

	//memory allocation

	hipMalloc((void **)&d_R, sizei);
	hipMalloc((void **)&d_G, sizei);
	hipMalloc((void **)&d_B, sizei);
	hipMalloc((void **)&d_Rnew, sizei);
	hipMalloc((void **)&d_Gnew, sizei);
	hipMalloc((void **)&d_Bnew, sizei);


	gettimeofday(&tim, NULL);
	double t2b=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("part 5b Allocation of device memory took :%.6lf seconds elapsed\n", t2b-t1b);
	//printf("%.6lf,", t2b-t1b );


	//5c : timing Transferring data between host and device mem
	gettimeofday(&tim, NULL);
	double t3=tim.tv_sec+(tim.tv_usec/1000000.0);

	hipMemcpy(d_R, R, sizei, hipMemcpyHostToDevice);
	hipMemcpy(d_G, G, sizei, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizei, hipMemcpyHostToDevice);
	gettimeofday(&tim, NULL);
	double t4=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("part 5c Transfer of data took :%.6lf seconds elapsed\n", t4-t3);
	//printf("%.6lf,", t4-t3 );


	//5d: time doing the blurring
	gettimeofday(&tim, NULL);
	double t5=tim.tv_sec+(tim.tv_usec/1000000.0);

	dim3 dimGrid(ceil(colsize/(float)TX), ceil(rowsize/(float)TY), 1);
	dim3 dimBlock(32, 32, 1);
	//run blurring
	for (k=0; k < nblurs; k++){
		blurKernel<<<dimGrid,dimBlock>>>(d_R, d_G, d_B, d_Rnew, d_Gnew, d_Bnew, rowsize, colsize);
		copyKernel<<<dimGrid,dimBlock>>>(d_R, d_G, d_B, d_Rnew, d_Gnew, d_Bnew, rowsize, colsize);
	}

	//return data back to host
	hipMemcpy(R, d_R, sizei, hipMemcpyDeviceToHost);
	hipMemcpy(G, d_G, sizei, hipMemcpyDeviceToHost);
	hipMemcpy(B, d_B, sizei, hipMemcpyDeviceToHost);

	hipFree(d_R); hipFree(d_G); hipFree(d_B);
	hipFree(d_Rnew); hipFree(d_Gnew); hipFree(d_Bnew);

	gettimeofday(&tim, NULL);
	double t6=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("part 5d Blurring took :%.6lf seconds elapsed\n", t6-t5);
	//printf("%.6lf,", t6-t5 );
	


	//5e: time to output blurred image
	gettimeofday(&tim, NULL);
	double t7=tim.tv_sec+(tim.tv_usec/1000000.0);

	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row][col],G[row][col],B[row][col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);
    //return 0;


	gettimeofday(&tim, NULL);
	double t8=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("part 5e Outputting blurre image took :%.6lf seconds elapsed\n", t8-t7);
	//printf("%.6lf,\n", t8-t7 );
	return 0;
}